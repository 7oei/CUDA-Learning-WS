#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;
    // Allocate memory
    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);
    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }
    
    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);
    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    // Executing kernel 
    vector_add<<<1,1>>>(d_out, d_a, d_b, N);
    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    printf("%f\n", out[0]);
    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    // Deallocate host memory
    free(a);
    free(b);
    free(out);
}